#include "hip/hip_runtime.h"
#pragma once

#include <stdint.h>
#include <cuda/std/utility>

#include "common.hpp"

template<unsigned W>
struct BitBoard {
  using state_t = std::conditional_t<W == 64, uint4, uint32_t>;
  using row_t = std::conditional_t<W == 64, uint64_t, uint32_t>;
  
  state_t state;

  _DI_ BitBoard() {
    if constexpr (W == 64) {
      state = {0, 0, 0, 0};
    } else {
      state = 0;
    }
  }
  
  _DI_ explicit BitBoard(state_t initial_state) : state(initial_state) {}
  
  _DI_ static BitBoard solid() {
    if constexpr (W == 64) {
      return BitBoard({~0U, ~0U, ~0U, ~0U});
    } else {
      return BitBoard(~0U);
    }
  }

  [[nodiscard]] _DI_ static BitBoard load(const row_t *data);
  _DI_ void save(row_t *data) const;

  _DI_ bool operator==(BitBoard other) const { return (*this ^ other).empty(); }

  _DI_ BitBoard operator~() const {
    if constexpr (W == 64) {
      return BitBoard({~state.x, ~state.y, ~state.z, ~state.w});
    } else {
      return BitBoard(~state);
    }
  }
  
  _DI_ BitBoard operator|(const BitBoard other) const {
    if constexpr (W == 64) {
      return BitBoard({state.x | other.state.x, state.y | other.state.y, state.z | other.state.z, state.w | other.state.w});
    } else {
      return BitBoard(state | other.state);
    }
  }
  
  _DI_ BitBoard operator&(const BitBoard other) const {
    if constexpr (W == 64) {
      return BitBoard({state.x & other.state.x, state.y & other.state.y, state.z & other.state.z, state.w & other.state.w});
    } else {
      return BitBoard(state & other.state);
    }
  }
  
  _DI_ BitBoard operator^(const BitBoard other) const {
    if constexpr (W == 64) {
      return BitBoard({state.x ^ other.state.x, state.y ^ other.state.y, state.z ^ other.state.z, state.w ^ other.state.w});
    } else {
      return BitBoard(state ^ other.state);
    }
  }
  
  _DI_ void operator|=(const BitBoard other) {
    if constexpr (W == 64) {
      state.x |= other.state.x; state.y |= other.state.y; state.z |= other.state.z; state.w |= other.state.w;
    } else {
      state |= other.state;
    }
  }
  
  _DI_ void operator&=(const BitBoard other) {
    if constexpr (W == 64) {
      state.x &= other.state.x; state.y &= other.state.y; state.z &= other.state.z; state.w &= other.state.w;
    } else {
      state &= other.state;
    }
  }
  
  _DI_ void operator^=(const BitBoard other) {
    if constexpr (W == 64) {
      state.x ^= other.state.x; state.y ^= other.state.y; state.z ^= other.state.z; state.w ^= other.state.w;
    } else {
      state ^= other.state;
    }
  }

  _DI_ row_t row(int y) const;
  _DI_ row_t column(int x) const;
  _DI_ bool get(int x, int y) const;
  _DI_ bool get(cuda::std::pair<int, int> cell) const { return get(cell.first, cell.second); }
  _DI_ void set(int x, int y);
  _DI_ void set(cuda::std::pair<int, int> cell) { set(cell.first, cell.second); }
  _DI_ void erase(int x, int y);
  _DI_ void erase(cuda::std::pair<int, int> cell) { erase(cell.first, cell.second); }

  _DI_ cuda::std::pair<int, int> first_on() const;

  _DI_ bool empty() const;
  _DI_ int pop() const;
};

template<unsigned W>
_DI_ BitBoard<W> BitBoard<W>::load(const row_t *in) {
  if constexpr (W == 64) {
    const uint4 *u4ptr = (const uint4 *)in;
    uint4 result = u4ptr[threadIdx.x & 31];
    return BitBoard(result);
  } else {
    return BitBoard(in[threadIdx.x & 31]);
  }
}

template<unsigned W>
_DI_ void BitBoard<W>::save(row_t *out) const {
  if constexpr (W == 64) {
    uint4 *u4ptr = (uint4 *)out;
    u4ptr[threadIdx.x & 31] = state;
  } else {
    out[threadIdx.x & 31] = state;
  }
}

template<unsigned W>
_DI_ typename BitBoard<W>::row_t BitBoard<W>::row(int y) const {
  if constexpr (W == 64) {
    int src = (y & 63) >> 1;

    if (y & 1) {
      uint32_t lo = __shfl_sync(0xffffffffu, state.z, src);
      uint32_t hi = __shfl_sync(0xffffffffu, state.w, src);
      return (uint64_t)hi << 32 | lo;
    } else {
      uint32_t lo = __shfl_sync(0xffffffffu, state.x, src);
      uint32_t hi = __shfl_sync(0xffffffffu, state.y, src);
      return (uint64_t)hi << 32 | lo;
    }
  } else {
    return __shfl_sync(0xffffffffu, state, y);
  }
}

template<unsigned W>
_DI_ typename BitBoard<W>::row_t BitBoard<W>::column(int x) const {
  if constexpr (W == 64) {
    uint32_t xs, zs;
    if(x < 32) {
      xs = __ballot_sync(0xffffffffu, state.x & (1<<x));
      zs = __ballot_sync(0xffffffffu, state.z & (1<<x));
    } else {
      xs = __ballot_sync(0xffffffffu, state.y & (1<<(x-32)));
      zs = __ballot_sync(0xffffffffu, state.w & (1<<(x-32)));
    }

    static const uint64_t B[] = {0x0000FFFF0000FFFF, 0x00FF00FF00FF00FF, 0x0F0F0F0F0F0F0F0F, 0x3333333333333333, 0x5555555555555555};
    static const unsigned S[] = {16, 8, 4, 2, 1};

    uint64_t xsl = xs;
    uint64_t zsl = zs;

    for(unsigned i = 0; i < sizeof(B)/sizeof(B[0]); i++) {
      xsl = (xsl | (xsl << S[i])) & B[i];
      zsl = (zsl | (zsl << S[i])) & B[i];
    }

    return xsl | (zsl << 1);
  } else {
    return __ballot_sync(0xffffffffu, state & (1<<x));
  }
}

template<unsigned W>
_DI_ bool BitBoard<W>::get(int x, int y) const {
  row_t r = row(y);
  return (r & ((row_t)1 << x)) != 0;
}

template<unsigned W>
_DI_ void BitBoard<W>::set(int x, int y) {
  if constexpr (W == 64) {
    bool should_act = (threadIdx.x & 31) == (y >> 1);
    unsigned int bit = 1u << (x & 31);

    state.x |= bit & (should_act && !(y & 1) && !(x & 32) ? 0xFFFFFFFF : 0);
    state.y |= bit & (should_act && !(y & 1) &&  (x & 32) ? 0xFFFFFFFF : 0);
    state.z |= bit & (should_act &&  (y & 1) && !(x & 32) ? 0xFFFFFFFF : 0);
    state.w |= bit & (should_act &&  (y & 1) &&  (x & 32) ? 0xFFFFFFFF : 0);
  } else {
    bool should_act = (threadIdx.x & 31) == y;
    unsigned int bit = 1u << (x & 31);

    if(should_act)
      state |= bit;
  }
}

template<unsigned W>
_DI_ void BitBoard<W>::erase(int x, int y) {
  if constexpr (W == 64) {
    bool should_act = (threadIdx.x & 31) == (y >> 1);
    unsigned int bit = 1u << (x & 31);

    state.x &= ~(bit & (should_act && !(y & 1) && !(x & 32) ? 0xFFFFFFFF : 0));
    state.y &= ~(bit & (should_act && !(y & 1) &&  (x & 32) ? 0xFFFFFFFF : 0));
    state.z &= ~(bit & (should_act &&  (y & 1) && !(x & 32) ? 0xFFFFFFFF : 0));
    state.w &= ~(bit & (should_act &&  (y & 1) &&  (x & 32) ? 0xFFFFFFFF : 0));
  } else {
    bool should_act = (threadIdx.x & 31) == y;
    unsigned int bit = 1u << (x & 31);

    if(should_act)
      state &= ~bit;
  }
}

template<unsigned W>
_DI_ cuda::std::pair<int, int> BitBoard<W>::first_on() const {
  if constexpr (W == 64) {
    int x_low = __ffsll((uint64_t) state.y << 32 | state.x) - 1;
    int x_high = __ffsll((uint64_t) state.w << 32 | state.z) - 1;

    bool use_high = ((state.x | state.y) == 0);
    int x = use_high ? x_high : x_low;

    int y_base = (threadIdx.x & 31) << 1;
    int y = y_base + (use_high ? 1 : 0);

    uint32_t mask = __ballot_sync(0xffffffffu, state.x | state.y | state.z | state.w);
    int first_lane = __ffs(mask) - 1;

    y = __shfl_sync(0xffffffff, y, first_lane);
    x = __shfl_sync(0xffffffff, x, first_lane);

    return {x, y};
  } else {
    int x = __ffsll(state) - 1;
    int y = threadIdx.x;

    uint32_t mask = __ballot_sync(0xffffffffu, state);
    int first_lane = __ffs(mask) - 1;

    y = __shfl_sync(0xffffffff, y, first_lane);
    x = __shfl_sync(0xffffffff, x, first_lane);

    return {x, y};
  }
}

template<unsigned W>
_DI_ bool BitBoard<W>::empty() const {
  if constexpr (W == 64) {
    return __ballot_sync(0xffffffffu, state.x | state.y | state.z | state.w) == 0;
  } else {
    return __ballot_sync(0xffffffffu, state) == 0;
  }
}

template<unsigned W>
_DI_ int BitBoard<W>::pop() const {
  int val;
  if constexpr (W == 64) {
    val = __popc(state.x) + __popc(state.y) + __popc(state.z) + __popc(state.w);
  } else {
    val = __popc(state);
  }
  for (int offset = 16; offset > 0; offset /= 2)
    val += __shfl_down_sync(0xffffffff, val, offset);
  return __shfl_sync(0xffffffff, val, 0);
}


