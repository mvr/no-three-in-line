#include "hip/hip_runtime.h"
#pragma once

#include "common.hpp"

#include "board.cu"

template <unsigned N, unsigned W>
struct ThreeBoard {
  BitBoard<W> knownOn;
  BitBoard<W> knownOff;

  _DI_ ThreeBoard() : knownOn{}, knownOff{} {}
  _DI_ explicit ThreeBoard(BitBoard<W> knownOn, BitBoard<W> knownOff) : knownOn{knownOn}, knownOff{knownOff} {}

  _DI_ bool operator==(ThreeBoard<N, W> other) const { return (knownOn == other.knownOn) && (knownOff == other.knownOff); }
  _DI_ bool operator!=(ThreeBoard<N, W> other) const { return !(*this == other); }

  static _DI_ BitBoard<W> bounds();
  static _DI_ BitBoard<W> line(cuda::std::pair<unsigned, unsigned> p, cuda::std::pair<unsigned, unsigned> q);

  _DI_ bool consistent() const;
  _DI_ unsigned unknown_pop() const;

  _DI_ ThreeBoard<N, W> force_orthogonal_horiz() const;
  _DI_ ThreeBoard<N, W> force_orthogonal_vert() const;
  _DI_ ThreeBoard<N, W> force_orthogonal() const { return force_orthogonal_horiz().force_orthogonal_vert(); }

  _DI_ BitBoard<W> eliminate_line(cuda::std::pair<unsigned, unsigned> p, cuda::std::pair<unsigned, unsigned> q);
  _DI_ void eliminate_all_lines(cuda::std::pair<unsigned, unsigned> p);
  _DI_ void eliminate_all_lines(BitBoard<W> seed);
  _DI_ void eliminate_all_lines() { eliminate_all_lines(knownOn); }
  _DI_ void propagate();

  template<Axis d>
  _DI_ void soft_branch(unsigned row);
  _DI_ void soft_branch_all();

  _DI_ cuda::std::pair<unsigned, unsigned> most_constrained_row() const;
  _DI_ cuda::std::pair<unsigned, unsigned> most_constrained_col() const;
  _DI_ cuda::std::pair<Axis, unsigned> most_constrained() const;
};

template <unsigned N, unsigned W>
_DI_ BitBoard<W> ThreeBoard<N, W>::bounds() {
  if constexpr (W == 64) {
    uint32_t row_bound_x = N >= 32 ? (~0) : (1 << N) - 1;
    uint32_t row_bound_y = N >= 32 ? (1 << (N - 32)) - 1 : 0;
    bool has_half = (threadIdx.x & 31) < ((N + 1) >> 1);
    bool has_full = (threadIdx.x & 31) < (N >> 1);
    BitBoard<W> result;
    result.state.x = has_half ? row_bound_x : 0;
    result.state.y = has_half ? row_bound_y : 0;
    result.state.z = has_full ? row_bound_x : 0;
    result.state.w = has_full ? row_bound_y : 0;
    return result;
  } else {
    uint32_t row_bound = N >= 32 ? (~0) : (1 << N) - 1;
    bool has_row = (threadIdx.x & 31) < N;
    BitBoard<W> result;
    result.state = has_row ? row_bound : 0;
    return result;
  }
}

template <unsigned N, unsigned W>
_DI_ bool ThreeBoard<N, W>::consistent() const {
  return (knownOn & knownOff).empty();
}

template <unsigned N, unsigned W>
_DI_ unsigned ThreeBoard<N, W>::unknown_pop() const {
  return N*N - (knownOn | knownOff).pop();
}

template <unsigned N, unsigned W>
_DI_ ThreeBoard<N, W> ThreeBoard<N, W>::force_orthogonal_horiz() const {
  ThreeBoard<N, W> result = *this;

  if constexpr (W == 64) {
    int on_pop_x = __popc(knownOn.state.x) + __popc(knownOn.state.y);
    if(on_pop_x == 2) {
      result.knownOff.state.x = ~knownOn.state.x;
      result.knownOff.state.y = ~knownOn.state.y;
    }
    if(on_pop_x > 2) {
      result.knownOn = BitBoard<W>::solid();
      result.knownOff = BitBoard<W>::solid();
    }

    int on_pop_z = __popc(knownOn.state.z) + __popc(knownOn.state.w);
    if(on_pop_z == 2) {
      result.knownOff.state.z = ~knownOn.state.z;
      result.knownOff.state.w = ~knownOn.state.w;
    }
    if(on_pop_z > 2) {
      result.knownOn = BitBoard<W>::solid();
      result.knownOff = BitBoard<W>::solid();
    }

    int off_pop_x = __popc(knownOff.state.x) + __popc(knownOff.state.y);
    if(off_pop_x == N - 2) {
      result.knownOn.state.x = ~knownOff.state.x;
      result.knownOn.state.y = ~knownOff.state.y;
    }
    if(off_pop_x > N - 2) {
      result.knownOn = BitBoard<W>::solid();
      result.knownOff = BitBoard<W>::solid();
    }

    int off_pop_z = __popc(knownOff.state.z) + __popc(knownOff.state.w);
    if(off_pop_z == N - 2) {
      result.knownOn.state.z = ~knownOff.state.z;
      result.knownOn.state.w = ~knownOff.state.w;
    }
    if(off_pop_z > N - 2) {
      result.knownOn = BitBoard<W>::solid();
      result.knownOff = BitBoard<W>::solid();
    }
  } else {
    int on_pop = __popc(knownOn.state);
    if(on_pop == 2) {
      result.knownOff.state = ~knownOn.state;
    }
    if(on_pop > 2) {
      result.knownOn = BitBoard<W>::solid();
      result.knownOff = BitBoard<W>::solid();
    }

    int off_pop = __popc(knownOff.state);
    if(off_pop == N - 2) {
      result.knownOn.state = ~knownOff.state;
    }
    if(off_pop > N - 2) {
      result.knownOn = BitBoard<W>::solid();
      result.knownOff = BitBoard<W>::solid();
    }
  }

  const BitBoard<W> bds = ThreeBoard<N, W>::bounds();
  result.knownOn &= bds;
  result.knownOff &= bds;

  return result;
}

struct BinaryCount {
  uint32_t bit0;
  uint32_t bit1;
  uint32_t overflow;

  _DI_ BinaryCount operator+(const BinaryCount other) const {
    const uint32_t out0 = bit0 ^ other.bit0;
    const uint32_t carry0 = bit0 & other.bit0;

    const uint32_t out1 = bit1 ^ other.bit1 ^ carry0;
    const uint32_t carry1 = (bit1 & other.bit1) | (carry0 & (bit1 | other.bit1));
    const uint32_t out_overflow = carry1 | overflow | other.overflow;

    return {out0, out1, out_overflow};
  }
  _DI_ void operator+=(const BinaryCount other) { *this = *this + other; };
};

_DI_ BinaryCount count_vertically(const uint32_t value) {
  BinaryCount result = {value, 0, 0};

  #pragma unroll
  for (int offset = 16; offset > 0; offset /= 2) {
    BinaryCount other;
    other.bit0 = __shfl_down_sync(0xffffffff, result.bit0, offset);
    other.bit1 = __shfl_down_sync(0xffffffff, result.bit1, offset);
    other.overflow = __shfl_down_sync(0xffffffff, result.overflow, offset);

    result += other;
  }

  result.bit0 = __shfl_sync(0xffffffff, result.bit0, 0);
  result.bit1 = __shfl_sync(0xffffffff, result.bit1, 0);
  result.overflow = __shfl_sync(0xffffffff, result.overflow, 0);

  return result;
}

template <unsigned N, unsigned W>
_DI_ ThreeBoard<N, W> ThreeBoard<N, W>::force_orthogonal_vert() const {
  ThreeBoard<N, W> result = *this;

  if constexpr (W == 64) {
    const BinaryCount on_count_xz = count_vertically(knownOn.state.x) + count_vertically(knownOn.state.z);
    const uint32_t on_count_xz_eq_2 = ~on_count_xz.overflow & on_count_xz.bit1 & ~on_count_xz.bit0;
    result.knownOff.state.x |= ~knownOn.state.x & on_count_xz_eq_2;
    result.knownOff.state.z |= ~knownOn.state.z & on_count_xz_eq_2;

    const uint32_t on_count_xz_gt_2 = on_count_xz.overflow | (on_count_xz.bit1 & on_count_xz.bit0);
    result.knownOn.state.x |= on_count_xz_gt_2;
    result.knownOff.state.x |= on_count_xz_gt_2;

    const BinaryCount on_count_yw = count_vertically(knownOn.state.y) + count_vertically(knownOn.state.w);
    const uint32_t on_count_yw_eq_2 = ~on_count_yw.overflow & on_count_yw.bit1 & ~on_count_yw.bit0;
    result.knownOff.state.y |= ~knownOn.state.y & on_count_yw_eq_2;
    result.knownOff.state.w |= ~knownOn.state.w & on_count_yw_eq_2;

    const uint32_t on_count_yw_gt_2 = on_count_yw.overflow | (on_count_yw.bit1 & on_count_yw.bit0);
    result.knownOn.state.y |= on_count_yw_gt_2;
    result.knownOff.state.y |= on_count_yw_gt_2;

    BitBoard<W> notKnownOff = ~knownOff & ThreeBoard<N, W>::bounds();

    const BinaryCount not_off_count_xz = count_vertically(notKnownOff.state.x) + count_vertically(notKnownOff.state.z);
    const uint32_t not_off_count_xz_eq_2 = ~not_off_count_xz.overflow & not_off_count_xz.bit1 & ~not_off_count_xz.bit0;
    result.knownOn.state.x |= ~knownOff.state.x & not_off_count_xz_eq_2;
    result.knownOn.state.z |= ~knownOff.state.z & not_off_count_xz_eq_2;

    const uint32_t not_off_count_xz_lt_2 = ~not_off_count_xz.overflow & ~not_off_count_xz.bit1;
    result.knownOn.state.x |= not_off_count_xz_lt_2;
    result.knownOff.state.x |= not_off_count_xz_lt_2;

    const BinaryCount not_off_count_yw = count_vertically(notKnownOff.state.y) + count_vertically(notKnownOff.state.w);
    const uint32_t not_off_count_yw_eq_2 = ~not_off_count_yw.overflow & not_off_count_yw.bit1 & ~not_off_count_yw.bit0;
    result.knownOn.state.y |= ~knownOff.state.y & not_off_count_yw_eq_2;
    result.knownOn.state.w |= ~knownOff.state.w & not_off_count_yw_eq_2;

    const uint32_t not_off_count_yw_lt_2 = ~not_off_count_yw.overflow & ~not_off_count_yw.bit1;
    result.knownOn.state.y |= not_off_count_yw_lt_2;
    result.knownOff.state.y |= not_off_count_yw_lt_2;
  } else {
    const BinaryCount on_count = count_vertically(knownOn.state);
    const uint32_t on_count_eq_2 = ~on_count.overflow & on_count.bit1 & ~on_count.bit0;
    result.knownOff.state |= ~knownOn.state & on_count_eq_2;

    const uint32_t on_count_gt_2 = on_count.overflow | (on_count.bit1 & on_count.bit0);
    result.knownOn.state |= on_count_gt_2;
    result.knownOff.state |= on_count_gt_2;

    BitBoard<W> notKnownOff = ~knownOff & ThreeBoard<N, W>::bounds();

    const BinaryCount not_off_count = count_vertically(notKnownOff.state);
    const uint32_t not_off_count_eq_2 = ~not_off_count.overflow & not_off_count.bit1 & ~not_off_count.bit0;
    result.knownOn.state |= ~knownOff.state & not_off_count_eq_2;

    const uint32_t not_off_count_lt_2 = ~not_off_count.overflow & ~not_off_count.bit1;
    result.knownOn.state |= not_off_count_lt_2;
    result.knownOff.state |= not_off_count_lt_2;
  }

  const BitBoard<W> bds = ThreeBoard<N, W>::bounds();
  result.knownOn &= bds;
  result.knownOff &= bds;

  return result;
}

template <unsigned N, unsigned W>
_DI_ BitBoard<W>
ThreeBoard<N, W>::eliminate_line(cuda::std::pair<unsigned, unsigned> p,
                              cuda::std::pair<unsigned, unsigned> q) {
  if (p.first == q.first || p.second == q.second)
    return BitBoard<W>();

  if (p.second > q.second)
    cuda::std::swap(p, q);

  cuda::std::pair<int, unsigned> delta = {(int)q.first - p.first, q.second - p.second};

  // TODO: lookup table
  int factor = binary_gcd(std::abs(delta.first), delta.second);
  delta.first = delta.first / factor;
  delta.second = delta.second / factor;

  bool smaller_oob = ((int)p.first < delta.first) || (p.second < delta.second);
  bool larger_oob = factor == 1
    && ((q.first + delta.first >= N) || (q.second + delta.second >= N));

  if(smaller_oob && larger_oob)
    return BitBoard<W>();

  unsigned p_quo = p.second / delta.second;
  unsigned p_rem = p.second % delta.second;

  BitBoard<W> result;

  if constexpr (W == 64) {
    {
      unsigned row = 2*threadIdx.x;
      if (row % delta.second == p_rem) {
        int col = p.first + ((int)(row / delta.second) - p_quo) * delta.first;
        if(col >= 0 && col < 32) result.state.x |= 1 << col;
        else if(col >= 32 && col < 64) result.state.y |= 1 << (col-32);
      }
      if (p.second == row || q.second == row) {
        result.state.x = 0;
        result.state.y = 0;
      }
    }

    {
      unsigned row = 2*threadIdx.x+1;
      if (row % delta.second == p_rem) {
        int col = p.first + ((int)(row / delta.second) - p_quo) * delta.first;
        if(col >= 0 && col < 32) result.state.z |= 1 << col;
        else if(col >= 32 && col < 64) result.state.w |= 1 << (col-32);
      }
      if (p.second == row || q.second == row) {
        result.state.z = 0;
        result.state.w = 0;
      }
    }
  } else {
    unsigned row = threadIdx.x;
    if (row % delta.second == p_rem) {
      int col = p.first + ((int)(row / delta.second) - p_quo) * delta.first;
      if(col >= 0 && col < 32) result.state |= 1 << col;
    }
    if (p.second == row || q.second == row) {
      result.state = 0;
    }
  }

  return result;
}

template <unsigned N, unsigned W>
_DI_ void
ThreeBoard<N, W>::eliminate_all_lines(cuda::std::pair<unsigned, unsigned> p) {
  BitBoard<W> qs = knownOn;
  for (auto q = qs.first_on(); !qs.empty();
       qs.erase(q), q = qs.first_on()) {
    knownOff |= eliminate_line(p, q);
  }
  knownOff &= bounds();
}

template <unsigned N, unsigned W>
_DI_ void
ThreeBoard<N, W>::eliminate_all_lines(BitBoard<W> seed) {
  for (auto p = seed.first_on(); !seed.empty();
       seed.erase(p), p = seed.first_on()) {
    eliminate_all_lines(p);
  }
  knownOff &= bounds();
}

template <unsigned N, unsigned W>
_DI_ void ThreeBoard<N, W>::propagate() {
  ThreeBoard<N, W> prev;

  BitBoard<W> doneOns = knownOn;

  do {
    prev = *this;

    ThreeBoard<N, W> prev2;
    do {
      prev2 = *this;
      *this = force_orthogonal();
      if(!consistent())
        return;
    } while(*this != prev2);

    eliminate_all_lines(knownOn & ~doneOns);
    doneOns = knownOn;
  } while (*this != prev);
}

template <unsigned N, unsigned W>
template <Axis d>
_DI_ void ThreeBoard<N, W>::soft_branch<d>(unsigned r) {
  auto row_knownOn = (d == Axis::Horizontal) ? knownOn.row(r) : knownOn.column(r);
  auto row_knownOff = (d == Axis::Horizontal) ? knownOff.row(r) : knownOff.column(r);
  
  unsigned on_count = (W == 64) ? __popcll(row_knownOn) : __popc(row_knownOn);
  if(on_count >= 2) return;

  unsigned off_count = (W == 64) ? __popcll(row_knownOff) : __popc(row_knownOff);
  unsigned unknown_count = N - on_count - off_count;
  
  if (on_count == 1 && unknown_count > SOFT_BRANCH_1_THRESHOLD) return;
  if (on_count == 0 && unknown_count > SOFT_BRANCH_2_THRESHOLD) return;

  ThreeBoard<N, W> common(BitBoard<W>::solid(), BitBoard<W>::solid());
  typename BitBoard<W>::row_t remaining = ~row_knownOn & ~row_knownOff & (((typename BitBoard<W>::row_t)1 << N) - 1);

  auto make_cell = [&](unsigned c) {
    return (d == Axis::Horizontal) ? cuda::std::pair<unsigned, unsigned>{c, r} : cuda::std::pair<unsigned, unsigned>{r, c};
  };

  auto first_bit = [](typename BitBoard<W>::row_t val) {
    return ((W == 64) ? __ffsll(val) : __ffs(val)) - 1;
  };

  auto try_placement = [&](auto cell) {
    ThreeBoard<N, W> subBoard = *this;
    subBoard.knownOn.set(cell);
    subBoard.eliminate_all_lines(cell);
    subBoard.propagate();
    
    if (!subBoard.consistent()) {
      knownOff.set(cell);
    } else {
      common.knownOn &= subBoard.knownOn;
      common.knownOff &= subBoard.knownOff;
    }
  };

  for (; remaining; remaining &= remaining - 1) {
    auto cell = make_cell(first_bit(remaining));

    if(on_count == 1) {
      try_placement(cell);
    } else {
      ThreeBoard<N, W> subBoard = *this;
      subBoard.knownOn.set(cell);

      auto row_knownOff2 = (d == Axis::Horizontal) ? subBoard.knownOff.row(r) : subBoard.knownOff.column(r);
      typename BitBoard<W>::row_t remaining2 = ~row_knownOff2 & (((typename BitBoard<W>::row_t)1 << N) - 1);

      for (; remaining2; remaining2 &= remaining2 - 1) {
        try_placement(make_cell(first_bit(remaining2)));
      }
    }
  }

  knownOn |= common.knownOn;
  knownOff |= common.knownOff;
}


// soft_branch_all implementation (combined)
template <unsigned N, unsigned W>
_DI_ void ThreeBoard<N, W>::soft_branch_all() {
  for (int r = 0; r < N; r++) {
    soft_branch<Axis::Horizontal>(r);
  }
  for (int r = 0; r < N; r++) {
    soft_branch<Axis::Vertical>(r);
  }
}

template <unsigned N, unsigned W>
_DI_ cuda::std::pair<unsigned, unsigned>
ThreeBoard<N, W>::most_constrained_row() const {
  if constexpr (W == 64) {
    BitBoard<W> known = knownOn | knownOff;
    unsigned unknown_xy = N - __popc(known.state.x) + __popc(known.state.y);
    unsigned unknown_zw = N - __popc(known.state.z) + __popc(known.state.w);

    if(knownOn.state.x == 0 && knownOn.state.y == 0)
      unknown_xy = unknown_xy * (unknown_xy - 1);

    if(knownOn.state.z == 0 && knownOn.state.w == 0)
      unknown_zw = unknown_zw * (unknown_zw - 1);

    if (threadIdx.x * 2 >= N || unknown_xy == 0)
      unknown_xy = std::numeric_limits<unsigned>::max();
    if (threadIdx.x * 2 + 1 >= N || unknown_zw == 0)
      unknown_zw = std::numeric_limits<unsigned>::max();

    unsigned row;
    unsigned unknown;

    if (unknown_xy < unknown_zw) {
      row = threadIdx.x * 2;
      unknown = unknown_xy;
    } else {
      row = threadIdx.x * 2 + 1;
      unknown = unknown_zw;
    }

    for (int offset = 16; offset > 0; offset /= 2) {
      unsigned other_row = __shfl_down_sync(0xffffffff, row, offset);
      unsigned other_unknown = __shfl_down_sync(0xffffffff, unknown, offset);
      if (other_unknown < unknown) {
        row = other_row;
        unknown = other_unknown;
      }
    }

    row = __shfl_sync(0xffffffff, row, 0);
    unknown = __shfl_sync(0xffffffff, unknown, 0);

    return {row, unknown};
  } else {
    BitBoard<W> known = knownOn | knownOff;
    unsigned unknown = N - __popc(known.state);

    if(knownOn.state == 0)
      unknown = unknown * (unknown - 1);

    if (threadIdx.x >= N || unknown == 0)
      unknown = std::numeric_limits<unsigned>::max();

    unsigned row = threadIdx.x;

    for (int offset = 16; offset > 0; offset /= 2) {
      unsigned other_row = __shfl_down_sync(0xffffffff, row, offset);
      unsigned other_unknown = __shfl_down_sync(0xffffffff, unknown, offset);
      if (other_unknown < unknown) {
        row = other_row;
        unknown = other_unknown;
      }
    }

    row = __shfl_sync(0xffffffff, row, 0);
    unknown = __shfl_sync(0xffffffff, unknown, 0);

    return {row, unknown};
  }
}

template <unsigned N, unsigned W>
_DI_ cuda::std::pair<unsigned, unsigned>
ThreeBoard<N, W>::most_constrained_col() const {
  unsigned best_col = 0;
  unsigned min_unknown = std::numeric_limits<unsigned>::max();

  for (unsigned c = 0; c < N; c++) {
    typename BitBoard<W>::row_t col_knownOn = knownOn.column(c);
    typename BitBoard<W>::row_t col_knownOff = knownOff.column(c);
    typename BitBoard<W>::row_t col_known = col_knownOn | col_knownOff;

    unsigned unknown;
    if constexpr (W == 64) {
      unknown = N - __popcll(col_known);
    } else {
      unknown = N - __popc(col_known);
    }

    if (col_knownOn == 0) {
      unknown = unknown * (unknown - 1);
    }

    if (unknown > 0 && unknown < min_unknown) {
      best_col = c;
      min_unknown = unknown;
    }
  }

  return {best_col, min_unknown};
}

template <unsigned N, unsigned W>
_DI_ cuda::std::pair<Axis, unsigned>
ThreeBoard<N, W>::most_constrained() const {
  auto [row, row_unknown] = most_constrained_row();
  auto [col, col_unknown] = most_constrained_col();
  if (row_unknown < col_unknown)
    return {Axis::Horizontal, row};
  else
    return {Axis::Vertical, col};
}
